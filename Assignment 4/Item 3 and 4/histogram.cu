#include "hip/hip_runtime.h"
 #include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdint.h>
#include <errno.h>
#include <assert.h>
#include <string.h>
#include <sys/io.h>
#include <cutil_inline.h> 
#include "histogram_kernel.cu"
#define HISTOGRAM64_BIN_COUNT 64
int N;
int ThreadsPerBlock;
int NumBlocks;
int *d_In;
int *p_hist;
int *g_hist;
int *atomic_timer;


void ParseArguments(int, char**);
__global__ void hist_gen(int* d_In,int* p_hist,int N, int tot_threads);
__global__ void merge_hist(int* p_hist, int* g_hist, int tot_par_hist, int tot_threads);
int main(int argc, char** argv)
{
	unsigned int timer_m = 0, timer_p = 0, timer_cpu = 0;
	cutilCheckError(cutCreateTimer(&timer_m));
	cutilCheckError(cutCreateTimer(&timer_p));
	cutilCheckError(cutCreateTimer(&timer_cpu));
    ParseArguments(argc, argv);
	int hist_cpu[HISTOGRAM64_BIN_COUNT],i;
	int a[N],final_hist[HISTOGRAM64_BIN_COUNT],times[NumBlocks];
	int tot_threads = ThreadsPerBlock*NumBlocks;
	int tot_par_hist;
	if(N>tot_threads)
		tot_par_hist = tot_threads;
	else
		tot_par_hist = N;
	
	srand(1);	// set rand() seed to 1 for repeatability 

	for(i=0;i<N;i++) {	// load array with digits
		  a[i] = rand() % HISTOGRAM64_BIN_COUNT;  // Specify the number to be 0-63
	}
	
	
	cutilCheckError(cutStartTimer(timer_cpu));
	for(i = 0; i < HISTOGRAM64_BIN_COUNT; i++){
		hist_cpu[i] = 0;
	}
	for(i = 0; i < N; i++){
		hist_cpu[a[i]]++;
	}
	cutilCheckError(cutStopTimer(timer_cpu));
	printf("CPU Histogram:\n");
        printf("\n....................................\n");
	for(i = 0; i < HISTOGRAM64_BIN_COUNT; i++){
		printf("%d  ",hist_cpu[i]);
	}
	printf("\n");
	
	cutilSafeCall( hipMalloc( (void **)&d_In, N*sizeof(int)) );
	//cutilSafeCall( hipMalloc( (void **)&p_hist, tot_par_hist*HISTOGRAM64_BIN_COUNT*sizeof(int)) );
	cutilSafeCall( hipMalloc( (void **)&g_hist, HISTOGRAM64_BIN_COUNT*sizeof(int)) );
	cutilSafeCall( hipMalloc( (void **)&atomic_timer, NumBlocks*sizeof(int)) );

	cutilCheckError(cutStartTimer(timer_m));
	hipMemcpy(d_In, a, N*sizeof(int), hipMemcpyHostToDevice);
	cutilCheckError(cutStopTimer(timer_m));
	cutilCheckError(cutStartTimer(timer_p));	

	hist_gen<<< NumBlocks, ThreadsPerBlock >>>(d_In, g_hist, N, tot_threads,atomic_timer);	
	cutilCheckError(cutStopTimer(timer_p));
	cutilCheckError(cutStartTimer(timer_m));
	hipMemcpy(final_hist, g_hist, HISTOGRAM64_BIN_COUNT*sizeof(int), hipMemcpyDeviceToHost);
	cutilCheckError(cutStopTimer(timer_m));

	hipMemcpy(times, atomic_timer, NumBlocks*sizeof(int), hipMemcpyDeviceToHost);

	printf("GPU Histogram: Total Thread=%d\n",tot_threads);
        printf("\n...................................................\n");
	for(i = 0; i < HISTOGRAM64_BIN_COUNT; i++){
		printf("%d  ",final_hist[i]);
	}
        printf(".................................................\n");
	printf("\nBlock times\n");
	for(i = 0; i < NumBlocks; i++){
		printf("%f  ",(double)times[i]/(double)CLOCKS_PER_SEC);
	}
	printf("\n");
	printf("Memory Transfer time: %f (ms) \n", cutGetTimerValue(timer_m));
	printf("Processing time: %f (ms) \n", cutGetTimerValue(timer_p));
	printf("Total GPU implementation time: %f (ms) \n", cutGetTimerValue(timer_p)+cutGetTimerValue(timer_m));
	printf("CPU implementation time: %f (ms) \n", cutGetTimerValue(timer_cpu));
	
	return 0;
}

void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--length") == 0 || strcmp(argv[i], "-length") == 0) {
            N = atoi(argv[i+1]);
	    i = i + 1;
        }
        if (strcmp(argv[i], "--threads") == 0 || strcmp(argv[i], "-threads") == 0) {
            ThreadsPerBlock = atoi(argv[i+1]);
	    i = i + 1;
        }
        if (strcmp(argv[i], "--blocks") == 0 || strcmp(argv[i], "-blocks") == 0) {
            NumBlocks = atoi(argv[i+1]);
	    i = i + 1;
         
        }
    }
}
