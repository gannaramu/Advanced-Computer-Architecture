#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdint.h>
#include <errno.h>
#include <assert.h>
#include <string.h>
#include <sys/io.h>
#include <cutil_inline.h> 
#include "histogram_kernel.cu"
#define HISTOGRAM64_BIN_COUNT 64

int N;
int ThreadsPerBlock;
int NumBlocks;
int *d_In;
int *p_hist;
int *g_hist;


void ParseArguments(int, char**);
__global__ void hist_gen(int* d_In,int* p_hist,int N, int tot_threads);
__global__ void merge_hist(int* p_hist, int* g_hist, int tot_par_hist, int tot_threads);


int main(int argc, char** argv)
{
/// Declaring all the timers !
	unsigned int timer_m = 0, timer_p = 0, timer_cpu = 0;
	cutilCheckError(cutCreateTimer(&timer_m));
	cutilCheckError(cutCreateTimer(&timer_p));
	cutilCheckError(cutCreateTimer(&timer_cpu));

        ParseArguments(argc, argv);
	
        int hist_cpu[HISTOGRAM64_BIN_COUNT],i;
	int a[N],final_hist[HISTOGRAM64_BIN_COUNT];
	int total_threads = ThreadsPerBlock*NumBlocks;
	int total_par_hist;

	if(N>total_threads)
		total_par_hist = total_threads;
	else
		total_par_hist = N;
	
	srand(1);	// set rand() seed to 1 for repeatability 

	for(i=0;i<N;i++) {	// load array with digits
		  a[i] = rand() % HISTOGRAM64_BIN_COUNT;  // Specify the number to be 0-63
	}
	
	
	cutilCheckError(cutStartTimer(timer_cpu));
	for(i = 0; i < HISTOGRAM64_BIN_COUNT; i++){
		hist_cpu[i] = 0;
	}
	for(i = 0; i < N; i++){
		hist_cpu[a[i]]++;
	}
	cutilCheckError(cutStopTimer(timer_cpu));

	printf("CPU Histogram:\n");
        printf(".......................................................\n");
	for(i = 0; i < HISTOGRAM64_BIN_COUNT; i++){
		printf("%d ",hist_cpu[i]);
               // printf("%d ",a[i]);
	}
	printf("\n..............................................................\n");
	

//// GPU Implementation starts
/// Mem allocation  
	cutilSafeCall( hipMalloc( (void **)&d_In, N*sizeof(int)) );
	cutilSafeCall( hipMalloc( (void **)&p_hist, total_par_hist * HISTOGRAM64_BIN_COUNT * sizeof(int)) );
	cutilSafeCall( hipMalloc( (void **)&g_hist, HISTOGRAM64_BIN_COUNT * sizeof(int)) );
	cutilCheckError(cutStartTimer(timer_m));
///// Mem transfer 
	hipMemcpy(d_In, a, N*sizeof(int), hipMemcpyHostToDevice);
//Kernel Call
	cutilCheckError(cutStopTimer(timer_m));
	cutilCheckError(cutStartTimer(timer_p));
	hist_gen<<< NumBlocks, ThreadsPerBlock >>>(d_In, p_hist, N, total_threads);
	cutilSafeCall( hipDeviceSynchronize() );
	merge_hist<<<2,32>>>(p_hist, g_hist, total_par_hist,total_threads);
	cutilCheckError(cutStopTimer(timer_p));
	cutilCheckError(cutStartTimer(timer_m));
/// Return transfer
	hipMemcpy(final_hist, g_hist, HISTOGRAM64_BIN_COUNT * sizeof(int), hipMemcpyDeviceToHost);
	cutilCheckError(cutStopTimer(timer_m));

// Printing 
	printf("GPU Histogram: part=%d\t, Total Number of Threads=%d\n",total_par_hist,total_threads);
        printf("\n.......................................................\n");

	for(i = 0; i < HISTOGRAM64_BIN_COUNT; i++){
		printf("%d ",final_hist[i]);
	}
printf("\n....................................\n");
	printf("\n");
	printf("Total Memory Transfer time: %f(ms) \n", cutGetTimerValue(timer_m));
	printf("GPU Processing time: %f(ms)\n", cutGetTimerValue(timer_p));
	printf("Total GPU Time: %f (ms) \n", cutGetTimerValue(timer_p)+cutGetTimerValue(timer_m));
	printf("CPU  Time: %f(ms)\n", cutGetTimerValue(timer_cpu));
	return 0;
}

void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--length") == 0 || strcmp(argv[i], "-length") == 0) {
            N = atoi(argv[i+1]);
	    i = i + 1;
        }
        if (strcmp(argv[i], "--threads") == 0 || strcmp(argv[i], "-threads") == 0) {
            ThreadsPerBlock = atoi(argv[i+1]);
	    i = i + 1;
        }
        if (strcmp(argv[i], "--blocks") == 0 || strcmp(argv[i], "-blocks") == 0) {
            NumBlocks = atoi(argv[i+1]);
	    i = i + 1;
         
        }
    }
}
